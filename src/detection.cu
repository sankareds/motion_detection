#include "hip/hip_runtime.h"
#include <stdio.h>
#include <iostream>
#include "opencv4/opencv2/imgproc.hpp"
#include <opencv4/opencv2/core/core.hpp>
#include <opencv4/opencv2/opencv.hpp>
#include <math.h>
#include <string>
#include <cuda/hip/hip_runtime.h>
#include <opencv4/opencv2/cudafeatures2d.hpp>
#include "opencv4/opencv2/imgproc/types_c.h"
#include "opencv4/opencv2/cudacodec.hpp"
#include "opencv4/opencv2/cudaimgproc.hpp"
#include "opencv4/opencv2/cudafilters.hpp"
#include "opencv4/opencv2/cudawarping.hpp"
#include <opencv4/opencv2/videoio.hpp>
#include <opencv4/opencv2/highgui.hpp>
#include <cuda/>

using namespace cv;
using namespace std;
using namespace cv::cuda;

__global__ void trainKernel(cuda::PtrStepSz<uchar>in_device,cuda::PtrStepSz<uchar>buffer, int k); //fill initial buffer
__global__ void calculateDiscr(cuda::PtrStepSz<uchar>in_device, cuda::PtrStepSz<uchar>buffer,cuda::PtrStepSz<uchar>disc_device, const int buff_size,cuda::PtrStepSz<uchar>old_disc_device); //calculate initial backgorund
__global__ void changeDetection(cuda::PtrStepSz<uchar>in_device, cuda::PtrStepSz<uchar>buffer_device, cuda::PtrStepSz<uchar>disc_device, cuda::PtrStepSz<uchar>out_device, cuda::PtrStepSz<uchar>k_device, int buff_size,cuda::PtrStepSz<uchar>old_disc_device); //motion detection
__device__ int sort_and_median(int arr[], int length); //calculate new dicriminator for a pixel


int main( int argc, char** argv )
{
    system("clear");
    int scale=1;
    const int buff_size=150; //buffer size in numer of frame
    int TRAIN=true; //flag for training fase
    hipEvent_t start, stop;
    float time;
    Mat img;
    cv::namedWindow("MyCameraPreview", cv::WINDOW_AUTOSIZE);
    const char* gst =  "rtspsrc location=rtsp://admin:@cam1/ch0_0.264 name=r latency=0 protocols=tcp ! application/x-rtp,payload=96,encoding-name=H264 ! rtph264depay ! h264parse ! nvv4l2decoder ! nvvidconv ! video/x-raw(memory:NVMM), format=BGRx ! nvvidconv ! videoconvert ! video/x-raw, format=BGR, framerate=5/1 ! appsink";
    cv::VideoCapture cap(gst, cv::CAP_GSTREAMER);
    if ( !cap.isOpened() )
    {
        cout << "Cannot open the video" << endl;
        return -1;
    }
    bool bSuccess = cap.read(img);
    if (!bSuccess)
    {

        cout << "Impossibile leggere frame di input" << endl;
        return 0;

    }
    Size s = img.size(); //calcolo dimensioni frame
    cout<<"Dimensioni originali "<<s.width<<" x "<<s.height<<endl;
    int N = s.height/scale;
    int M = s.width/scale;
    cout<<"Dimensioni elaborazione "<<M<<" x "<<N<<endl;


    cv::VideoWriter oVW ("output.avi", cv::VideoWriter::fourcc('M','P','4','2'), 5, Size(M,N), false); //inizializza oggetto VideoWriter
    if ( !oVW.isOpened() ) //if not initialize the VideoWriter successfully, exit the program
    {
        cout << "ERROR: Failed to write the video" << endl;
        return -1;
    }

	cv::VideoWriter bVW ("background.avi", cv::VideoWriter::fourcc('M','P','4','2'), 5, Size(M,N), false); //inizializza oggetto VideoWriter
	if ( !bVW.isOpened() ) //if not initialize the VideoWriter successfully, exit the program
    {
        cout << "ERROR: Failed to write the video" << endl;
        return -1;
    }
    Mat inframe; //input frame
    Mat outframe(N, M, CV_8UC1, Scalar(0)); //output frame
    Mat disc(N, M, CV_8UC3, Scalar(0,0,0)); //background
    Mat old_disc(N, M, CV_8UC3, Scalar(0,0,0)); //old backgorund for comparison
    Mat resized (N, M, CV_8UC3, Scalar(0,0,0)); //resized image
    Mat k(N, M, CV_8UC1,Scalar(0)); //last buffer element pointer
    Mat buffer(N*buff_size,M, CV_8UC3 , Scalar(0,0,0)); //buffer for motion memorization
    Mat element = getStructuringElement( MORPH_RECT, Size(3, 3), Point( 1, 1) );
    cuda::GpuMat in_device, out_device, disc_device, buffer_device, k_device, old_disc_device, resized_device; //same Mats on Gpu
	//GpuMats upload on Gpu
    disc_device.upload(disc);
    old_disc_device.upload(old_disc);
    resized_device.upload(resized);
    buffer_device.upload(buffer);
    k_device.upload(k);
    int N_FRAME=1; // frame counter
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord( start, 0 );
    while(1)
    {
        outframe.setTo(0);
        out_device.upload(outframe);
        if (TRAIN==true) // training phase for initial background
        {
            cout<<"Training sfondo"<<endl;
            for(int y=0; y<buff_size; y++)
            {
                    bool bSuccess = cap.read(inframe);
                    if (!bSuccess)
                    {
                        cout << "Impossibile leggere frame di input" << endl;
                        return -1;

                    }
                    if (inframe.empty()) //check whether the image is loaded or not
                    {
                        cout << "Cannot open frame for train" << endl;
                        return -2;
                    }
                    N_FRAME++;
                    in_device.upload(inframe);
                    cuda::resize(in_device, resized_device, Size(M,N));
                    cv::Ptr<cv::cuda::Filter> filter = cuda::createGaussianFilter(resized_device.type(), resized_device.type(), Size(3, 3),0);
                                filter->apply(resized_device, resized_device);
                    filter -> apply(resized_device, resized_device);
                    trainKernel<<<M, N>>>(resized_device, buffer_device, y);
            }
            cout<<"Calcolo sfondo iniziale"<<endl;
            calculateDiscr<<<M, N>>>(resized_device, buffer_device, disc_device, buff_size, old_disc_device);
            cout<<"Elaborazione"<<endl;
            TRAIN=false;
        }
        else
        {
            bool bSuccesss = cap.read(inframe);
            if (!bSuccesss)
            {
                cout << "Impossibile leggere frame di input" << endl;
                hipEventRecord( stop, 0 );
                hipEventSynchronize( stop );
                hipEventElapsedTime( &time, start, stop );
                printf("execution time %8.2f ms\nAvg. FPS: %8.2f for %d frame\n",time, N_FRAME/(time/1000), N_FRAME);
                hipEventDestroy( start );
                hipEventDestroy( stop );
                return -1;

            }
            if (inframe.empty()) //check whether the image is loaded or not
            {
                cout << "Video ended" << endl;
                hipEventRecord( stop, 0 );
                hipEventSynchronize( stop );
                hipEventElapsedTime( &time, start, stop );
                printf("execution time %8.2f ms\nAvg. FPS: %8.2f for %d frame\n",time, N_FRAME/(time/1000), N_FRAME);
                hipEventDestroy( start );
                hipEventDestroy( stop );
                return -1;
            }
            in_device.upload(inframe); //carico frame input su GPU
            cuda::resize(in_device, resized_device, Size(M,N));
            cv::Ptr<cv::cuda::Filter> filter = cuda::createGaussianFilter(resized_device.type(), resized_device.type(), Size(3, 3),0);
            filter->apply(resized_device, resized_device);
            changeDetection<<<M, N>>>(resized_device, buffer_device, disc_device, out_device, k_device, buff_size, old_disc_device);
            filter = cuda::createMorphologyFilter(CV_MOP_OPEN, out_device.type(), element);
            filter->apply(out_device, out_device);
            Mat result_host(out_device);
            cout << "cols=" << disc_device.cols << endl;
            cout << "rows=" << disc_device.rows << endl;
            Mat back_host(disc_device);
            cv::imshow("MyCameraPreview",result_host);
    		if((char)cv::waitKey(1) == (char)27)
    			break;
            oVW.write(result_host);
            bVW.write(back_host);
            N_FRAME++;
        }
    }
    return 0;
}


__global__ void trainKernel(cuda::PtrStepSz<uchar>in_device,cuda::PtrStepSz<uchar>buffer, int k)
{
    int y=threadIdx.x;
    int x=blockIdx.x;
    buffer.ptr(y+k*in_device.rows)[x*3]=in_device.ptr(y)[x*3];
    buffer.ptr(y+k*in_device.rows)[x*3+1]=in_device.ptr(y)[x*3+1];
    buffer.ptr(y+k*in_device.rows)[x*3+2]=in_device.ptr(y)[x*3+2];
    __syncthreads();
}

__global__ void calculateDiscr(cuda::PtrStepSz<uchar>in_device, cuda::PtrStepSz<uchar>buffer_device,cuda::PtrStepSz<uchar>disc_device, const int buff_size,cuda::PtrStepSz<uchar>old_disc_device)
{
    int y=threadIdx.x;
    int x=blockIdx.x;
    int arrR[150];
    int arrG[150];
    int arrB[150];
    //scorro lista per calcolare il giusto discriminatore per ogni pixel
    for(int k=0; k<buff_size; k++)
    {
        // il discrminatore é dato dalla media dei valori RGB del pixel
        arrR[k]=buffer_device.ptr(y+k*in_device.rows)[x*3+2];
        arrG[k]=buffer_device.ptr(y+k*in_device.rows)[x*3+1];
        arrB[k]=buffer_device.ptr(y+k*in_device.rows)[x*3];
    }
    disc_device.ptr(y)[x*3+2]=sort_and_median(arrR, buff_size);
    disc_device.ptr(y)[x*3+1]=sort_and_median(arrG, buff_size);
    disc_device.ptr(y)[x*3]=sort_and_median(arrB, buff_size);
    old_disc_device.ptr(y)[x*3+2]=disc_device.ptr(y)[x*3+2];
    old_disc_device.ptr(y)[x*3+1]=disc_device.ptr(y)[x*3+1];
    old_disc_device.ptr(y)[x*3]=disc_device.ptr(y)[x*3];
    __syncthreads();
}

__global__ void changeDetection(cuda::PtrStepSz<uchar>in_device, cuda::PtrStepSz<uchar>buffer_device, cuda::PtrStepSz<uchar>disc_device, cuda::PtrStepSz<uchar>out_device, cuda::PtrStepSz<uchar>k_device, int buff_size,cuda::PtrStepSz<uchar>old_disc_device)
{
    int y=threadIdx.x;
    int x=blockIdx.x;
    int R_diff= abs(disc_device.ptr(y)[x*3+2]-in_device.ptr(y)[x*3+2]);
    int G_diff= abs(disc_device.ptr(y)[x*3+1]-in_device.ptr(y)[x*3+1]);
    int B_diff= abs(disc_device.ptr(y)[x*3]-in_device.ptr(y)[x*3]);
    int Sim=(R_diff+G_diff+B_diff)/3;
    if(Sim>=10)  //   if movement
    {
        int R_old_diff= abs(old_disc_device.ptr(y)[x*3+2]-in_device.ptr(y)[x*3+2]);
        int G_old_diff= abs(old_disc_device.ptr(y)[x*3+1]-in_device.ptr(y)[x*3+1]);
        int B_old_diff= abs(old_disc_device.ptr(y)[x*3]-in_device.ptr(y)[x*3]);
        int old_Sim=(R_old_diff+G_old_diff+B_old_diff)/3;
        if(old_Sim<=10)  //if movement less than the old
        {
            disc_device.ptr(y)[x*3+2]=old_disc_device.ptr(y)[x*3+2];
            disc_device.ptr(y)[x*3+1]=old_disc_device.ptr(y)[x*3+1];
            disc_device.ptr(y)[x*3]=old_disc_device.ptr(y)[x*3];
            k_device.ptr(y)[x]=0;
        }
        else
        {
            out_device.ptr(y)[x]=255;
            if(k_device.ptr(y)[x]>=buff_size)
            {
                int arrR[150];
                int arrG[150];
                int arrB[150];
                //scroll through the list to calculate the right discriminator for each pixel
                for(int k=0; k<buff_size; k++)
                {
                    // the discriminator is given by the average of the RGB values ​​of the pixel
                    arrR[k]=buffer_device.ptr(y+k*in_device.rows)[x*3+2];
                    arrG[k]=buffer_device.ptr(y+k*in_device.rows)[x*3+1];
                    arrB[k]=buffer_device.ptr(y+k*in_device.rows)[x*3];
                }
                old_disc_device.ptr(y)[x*3+2]=disc_device.ptr(y)[x*3+2];
                old_disc_device.ptr(y)[x*3+1]=disc_device.ptr(y)[x*3+1];
                old_disc_device.ptr(y)[x*3]=disc_device.ptr(y)[x*3];

                disc_device.ptr(y)[x*3+2]=sort_and_median(arrR, buff_size);
                disc_device.ptr(y)[x*3+1]=sort_and_median(arrG, buff_size);
                disc_device.ptr(y)[x*3]=sort_and_median(arrB, buff_size);
                k_device.ptr(y)[x]=0;
            }
            else
            {
                int k=k_device.ptr(y)[x];
                buffer_device.ptr(y+k*in_device.rows)[x*3+2]=in_device.ptr(y)[x*3+2];
                buffer_device.ptr(y+k*in_device.rows)[x*3+1]=in_device.ptr(y)[x*3+1];
                buffer_device.ptr(y+k*in_device.rows)[x*3]=in_device.ptr(y)[x*3];
                k_device.ptr(y)[x]++;
            }
        }
    }
    else
    {
        k_device.ptr(y)[x]=0;
    }
    __syncthreads();
}



__device__ int sort_and_median(int arr[], int length)
{
    int i, j, tmp;
    for (i = 1; i < length; i++)
    {
        j = i;
        while (j > 0 && arr[j - 1] > arr[j])
        {
            tmp = arr[j];
            arr[j] = arr[j - 1];
            arr[j - 1] = tmp;
            j--;
        }
    }
    return arr[length/2];
}
